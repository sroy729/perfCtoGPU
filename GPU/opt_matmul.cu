#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#ifndef SIZE
#define SIZE 1024  // Default matrix size if not specified with -DSIZE=
#endif

#define TILE_SIZE 32  // 32x32 tiling for shared memory

__global__ void matMulKernel(const float* __restrict__ A,
                             const float* __restrict__ B,
                             float* __restrict__ C,
                             int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float Cvalue = 0.0f;

    #pragma unroll
    for (int t = 0; t < N / TILE_SIZE; ++t) {
        As[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            Cvalue += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    C[row * N + col] = Cvalue;
}

void matrixMultiplyGPU(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(N / TILE_SIZE, N / TILE_SIZE);

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Execution Time: %.3f ms\n", milliseconds);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void fillMatrix(float* mat, int N) {
    for (int i = 0; i < N * N; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    int N = SIZE;
    size_t size = N * N * sizeof(float);

    float* A = (float*)malloc(size);
    float* B = (float*)malloc(size);
    float* C = (float*)malloc(size);

    srand(time(NULL));
    fillMatrix(A, N);
    fillMatrix(B, N);

    matrixMultiplyGPU(A, B, C, N);

    // Optionally verify or print part of the result
    printf("C[0][0] = %f\n", C[0]);

    free(A);
    free(B);
    free(C);

    return 0;
}

