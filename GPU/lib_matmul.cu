#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

#ifndef SIZE
#define SIZE 1024  // Default matrix size
#endif

void matrixMultiply_cuBLAS(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    // Allocate memory on device
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // CUDA event for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Matrix multiplication: C = alpha * A * B + beta * C
    // cuBLAS uses column-major format, so we pass d_B before d_A
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, N, N,
                &alpha,
                d_B, N,
                d_A, N,
                &beta,
                d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Execution Time: " << milliseconds << " ms\n";

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void fillMatrix(float* mat, int N) {
    for (int i = 0; i < N * N; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    int N = SIZE;
    size_t size = N * N * sizeof(float);

    float* A = (float*)malloc(size);
    float* B = (float*)malloc(size);
    float* C = (float*)malloc(size);

    srand(static_cast<unsigned int>(time(nullptr)));
    fillMatrix(A, N);
    fillMatrix(B, N);

    matrixMultiply_cuBLAS(A, B, C, N);

    std::cout << "C[0][0] = " << C[0] << std::endl;

    free(A);
    free(B);
    free(C);

    return 0;
}

